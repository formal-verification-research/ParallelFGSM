
#include <hip/hip_runtime.h>
#include <hipblas.h>

__global__ void doublify(float *a)
{
  int idx = threadIdx.x + threadIdx.y*4;
  a[idx] *= 2;
}
